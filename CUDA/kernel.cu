#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <Windows.h>
#include <iostream>
#include <ctime>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
__global__ void multKernel(int *c, const int *a, const int *b, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i<n) c[i] = a[i] * b[((i * 1) % n + n) % n];
}


//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}

//int main(int argc, char *  argv[])
//{
//	int n = 10000000;
//	int *a = new int[n];
//	int *b = new int[n];
//	int *c = new int[n];
//	int t = GetTickCount();
//	for (int i = 0; i<n; i++)
//	{
//		a[i] = rand();
//		b[i] = rand();
//	}
//	for (int i = 0; i<n; i++)
//	{
//		c[i] = a[i] * b[i];
//		// printf("c = %i,	a = %i,	b = %i\n", c[i], a[i], b[i]); 
//	}
//
//	std::cout << "Time = " << t << std::endl;
//	system("pause");
//}

int main(int argc, char * argv[])
{
	int t = GetTickCount();
	int  n = 10000000;
	int *a = new int[n];
	int *b = new int[n];
	int *c = new int[n];
	int *d = new int[n];
	int *_a;
	int *_b;
	int *_c;

	for (int i = 0; i<n; i++)
	{
		a[i] = rand();
		b[i] = rand();
	}
	int startTime = clock();
	for (int i = 0; i < n; i++)
	{
		c[i] = a[i] * b[((i * 1) % n + n )%n];
	}
	int endTime = clock();
	std::cout << "Run time on CPU = " << endTime - startTime << std::endl;

	if (hipMalloc((void**)&_a, n * sizeof(int)) != hipSuccess) std::cout << "Error CudaMalloc1";
	if (hipMalloc((void**)&_b, n * sizeof(int)) != hipSuccess) std::cout << "Error CudaMalloc2";
	if (hipMalloc((void**)&_c, n * sizeof(int)) != hipSuccess) std::cout << "Error CudaMalloc3";

	if (hipMemcpy(_a, a, n * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) std::cout << "Error CudaMemcpy4" << std::endl;
	if (hipMemcpy(_b, b, n * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) std::cout << "Error CudaMemcpy5" << std::endl;

	startTime = clock();
	multKernel << <n / 512 + 1, 512 >> >(_c, _a, _b, n);
	if (hipDeviceSynchronize() != hipSuccess) std::cout << "Error CudaDeviceSynchronize7";
	endTime = clock();
	std::cout << "Run time on GPU = " << endTime - startTime << std::endl;
	
	if (hipMemcpy(d, _c, n * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) std::cout << "Error  CudaMemcpy6" << std::endl;
	for (int i = 0; i < n; i++)
	{
		if (c[i] != d[i]) std::cout << "Arrays are not equal : " << c[i] << " != " << d[i] << std::endl;
	}
	std::system("pause");
}


//int main(int argc, char *  argv[]) {
//	int		deviceCount;
//	hipDeviceProp_t	devProp;
//	hipGetDeviceCount(&deviceCount);
//	printf("Found %d devices\n", deviceCount);
//	for (int device = 0; device < deviceCount; device++) {
//		hipGetDeviceProperties(&devProp, device);
//		printf("Device %d\n", device);
//		printf("Compute capability     : %d.%d\n", devProp.major, devProp.minor);
//		printf("Name                   : %s\n", devProp.name);
//		printf("Total Global Memory    : %d\n", devProp.totalGlobalMem);
//		printf("Shared memory per block: %d\n", devProp.sharedMemPerBlock);
//		printf("Registers per block    : %d\n", devProp.regsPerBlock);
//		printf("Warp size              : %d\n", devProp.warpSize);
//		printf("Max threads per block  : %d\n", devProp.maxThreadsPerBlock);
//		printf("Total constant memory  : %d\n", devProp.totalConstMem);
//	}
//	return 0;
//}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
