#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <Windows.h>
#include <iostream>
#include <ctime>
#include <string>

#include "Hash.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
__global__ void multKernel(int *c, const int *a, const int *b, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i<n) c[i] = a[i] * b[((i * 1) % n + n) % n];
}

//int main(int argc, char * argv[])
//{
//	int t = GetTickCount();
//	int  n = 10000000;
//	int *a = new int[n];
//	int *b = new int[n];
//	int *c = new int[n];
//	int *d = new int[n];
//	int *_a;
//	int *_b;
//	int *_c;
//
//	for (int i = 0; i<n; i++)
//	{
//		a[i] = rand();
//		b[i] = rand();
//	}
//	int startTime = clock();
//	for (int i = 0; i < n; i++)
//	{
//		c[i] = a[i] * b[((i * 1) % n + n )%n];
//	}
//	int endTime = clock();
//	std::cout << "Run time on CPU = " << endTime - startTime << std::endl;
//
//	if (hipMalloc((void**)&_a, n * sizeof(int)) != hipSuccess) std::cout << "Error CudaMalloc1";
//	if (hipMalloc((void**)&_b, n * sizeof(int)) != hipSuccess) std::cout << "Error CudaMalloc2";
//	if (hipMalloc((void**)&_c, n * sizeof(int)) != hipSuccess) std::cout << "Error CudaMalloc3";
//
//	if (hipMemcpy(_a, a, n * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) std::cout << "Error CudaMemcpy4" << std::endl;
//	if (hipMemcpy(_b, b, n * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) std::cout << "Error CudaMemcpy5" << std::endl;
//
//	startTime = clock();
//	multKernel << <n / 512 + 1, 512 >> >(_c, _a, _b, n);
//	if (hipDeviceSynchronize() != hipSuccess) std::cout << "Error CudaDeviceSynchronize7";
//	endTime = clock();
//	std::cout << "Run time on GPU = " << endTime - startTime << std::endl;
//	
//	if (hipMemcpy(d, _c, n * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) std::cout << "Error  CudaMemcpy6" << std::endl;
//	for (int i = 0; i < n; i++)
//	{
//		if (c[i] != d[i]) std::cout << "Arrays are not equal : " << c[i] << " != " << d[i] << std::endl;
//	}
//	std::system("pause");
//}

int main(int argc, char *argv[])
{
	hash::Hash hash;
	
	std::string str = "Appolon13";
	/*std::cout << "Type string you want to hash" << std::endl;
	std::cin >> str;
	std::cout << "Type length of hash string" << std::endl;
	unsigned int length;
	std::cin >> length;*/
	str = hash.GetHash(str, 5);
	std::cout << str << std::endl;

	system("PAUSE");
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
