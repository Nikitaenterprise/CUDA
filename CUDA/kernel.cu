#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <Windows.h>
#include <iostream>
#include <ctime>
#include <string>

#include "Hash.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
__global__ void multKernel(int *c, const int *a, const int *b, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i<n) c[i] = a[i] * b[((i * 1) % n + n) % n];
}
__global__ void multMatrix(float *c, const float *a, const float *b, int m)
{
	int   bx = blockIdx.x;
	int   by = blockIdx.y;
	int   tx = threadIdx.x;
	int   ty = threadIdx.y;
	float sum = 0.0f;
	int   ia = m * 16 * by + m * ty;
	int   ib = 16 * bx + tx;
	int   ic = m * 16 * by + 16 * bx;

	for (int k = 0; k < m; k++)
		sum += a[ia + k] * b[ib + k*m];

	c[ic + m * ty + tx] = sum;
}

//int main(int argc, char * argv[])
//{
//	int t = GetTickCount();
//	int  n = 10000000;
//	int *a = new int[n];
//	int *b = new int[n];
//	int *c = new int[n];
//	int *d = new int[n];
//	int *_a;
//	int *_b;
//	int *_c;
//
//	for (int i = 0; i<n; i++)
//	{
//		a[i] = rand();
//		b[i] = rand();
//	}
//	int startTime = clock();
//	for (int i = 0; i < n; i++)
//	{
//		c[i] = a[i] * b[((i * 1) % n + n )%n];
//	}
//	int endTime = clock();
//	std::cout << "Run time on CPU = " << endTime - startTime << std::endl;
//
//	if (hipMalloc((void**)&_a, n * sizeof(int)) != hipSuccess) std::cout << "Error CudaMalloc1";
//	if (hipMalloc((void**)&_b, n * sizeof(int)) != hipSuccess) std::cout << "Error CudaMalloc2";
//	if (hipMalloc((void**)&_c, n * sizeof(int)) != hipSuccess) std::cout << "Error CudaMalloc3";
//
//	if (hipMemcpy(_a, a, n * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) std::cout << "Error CudaMemcpy4" << std::endl;
//	if (hipMemcpy(_b, b, n * sizeof(int), hipMemcpyHostToDevice) != hipSuccess) std::cout << "Error CudaMemcpy5" << std::endl;
//
//	startTime = clock();
//	multKernel <<<n / 512 + 1, 512 >>> (_c, _a, _b, n);
//	if (hipDeviceSynchronize() != hipSuccess) std::cout << "Error CudaDeviceSynchronize7";
//	endTime = clock();
//	std::cout << "Run time on GPU = " << endTime - startTime << std::endl;
//	
//	if (hipMemcpy(d, _c, n * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) std::cout << "Error  CudaMemcpy6" << std::endl;
//	for (int i = 0; i < n; i++)
//	{
//		if (c[i] != d[i]) std::cout << "Arrays are not equal : " << c[i] << " != " << d[i] << std::endl;
//	}
//
//	float *X, *Y, *result, *result1;
//	int m = 1024;
//	X = new float[m*m];
//	Y = new float[m*m];
//	result = new float[m*m];
//	result1 = new float[m*m];
//	float *_X, *_Y, *_result;
//	srand(235806);
//	for (int i = 0; i < m; i++)
//	{
//		for (int j = 0; j < m; j++)
//		{
//			X[m*i+j] = rand()%500;
//			Y[m*i+j] = rand()%500;
//		}
//	}
//	startTime = clock();
//	for (int i = 0; i < m; i++)
//	{
//		for (int j = 0; j < m; j++)
//		{
//			result[m*i + j] = 0;
//			for (int k = 0; k < m; k++)
//			{
//				result[m*i + j] += X[i*m + k] * Y[k*m + j];
//			}
//		}
//	}
//	endTime = clock();
//	std::cout << "Calculating on CPU = " << endTime - startTime << std::endl;
//	if (hipMalloc((void**)&_X, m*m * sizeof(float)) != hipSuccess) std::cout << "Error in first malloc" << std::endl;
//	if (hipMalloc((void**)&_Y, m*m * sizeof(float)) != hipSuccess) std::cout << "Error in second malloc" << std::endl;
//	if (hipMalloc((void**)&_result, m*m * sizeof(float)) != hipSuccess) std::cout << "Error in third malloc" << std::endl;
//	if (hipMemcpy(_X, X, m*m * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) std::cout << "Error in first copy" << std::endl;
//	if (hipMemcpy(_Y, Y, m*m * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) std::cout << "Error in second copy" << std::endl;
//	dim3 threads(16, 16);
//	dim3 blocks(m / threads.x, m / threads.y);
//	startTime = clock();
//	multMatrix << <blocks, threads >> > (_result, _X, _Y, m);
//	hipDeviceSynchronize();
//	if (hipDeviceSynchronize() != hipSuccess) std::cout << "Error in syncronization" << std::endl;
//	endTime = clock();
//	std::cout << "Calculating on GPU = " << endTime - startTime << std::endl;
//	if (hipMemcpy(result1, _result, m*m * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) std::cout << "Error in third copy" << std::endl;
//	for (int i = 0; i < m; i++)
//	{
//		for (int j = 0; j < m; j++)
//		{
//			if (abs(result[m*i + j] - result1[m*i + j]) > 0.00001)
//			{
//				std::cout << "n*i+j = " << m*i + j << "\t" << "result[n*i + j] = " << result[m*i + j] << "\t" << "result1[n*i + j] = " << result1[m*i + j] << std::endl;
//			}
//		}
//	}
//	std::system("pause");
//}

int main(int argc, char *argv[])
{
	hash::Hash hash;
	
	std::string str = "Zdorov";
	/*std::cout << "Type string you want to hash" << std::endl;
	std::cin >> str;
	std::cout << "Type length of hash string" << std::endl;
	unsigned int length;
	std::cin >> length;*/
	str = hash.GetHash(str, 10);
	std::cout << str << std::endl;

	system("PAUSE");
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
